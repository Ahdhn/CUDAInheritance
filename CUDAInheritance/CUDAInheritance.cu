#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "gtest/gtest.h"

#include "helper.h"

class Base
{
   public:
    __device__ __host__ Base() : m_data(0)
    {
    }

    __device__ __host__ __forceinline__ int get_data() const
    {
        return m_data;
    }

    __device__ __host__ __forceinline__ void set_data1(int d)
    {
        m_data = d;
    }

    virtual ~Base() = default;

    int m_data;
};

class Derived : public Base
{
   public:
    __device__ __host__ Derived(){};

    __device__ __host__ __forceinline__ int get_data() const
    {
        return this->m_data;
    };

    __device__ __host__ __forceinline__ void set_data(int d, bool base)
    {
        if (base) {
            this->set_data1(d);
        } else {
            m_d = d;
        }
    };

    virtual ~Derived() = default;

    int m_d;
};

__global__ void kernel(Derived d, const int data, const bool base)
{
    d.set_data(data, base);
}

TEST(CUDAInheritance, Test0)
{
    Derived      d;
    int          val = rand();
    int          num_run = 1E6;
    hipStream_t stream;
    CUDA_ERROR(hipStreamCreate(&stream));

    std::cout << "Accessing base's method through derived class --- ";
    CUDATimer timer;
    timer.start(stream);
    for (int n = 0; n < num_run; ++n) {
        kernel<<<1, 1, 0, stream>>>(d, val, true);
    }
    timer.stop();
    std::cout << " time = " << timer.elapsed_millis() << " (ms)\n";

    CUDA_ERROR(hipDeviceSynchronize());

    std::cout << "Accessing only derived class methods --- ";
    val = rand();
    timer.start(stream);
    for (int n = 0; n < num_run; ++n) {
        kernel<<<1, 1, 0, stream>>>(d, val, false);
    }
    timer.stop();
    std::cout << " time = " << timer.elapsed_millis() << " (ms)\n";

    hipError_t status = hipDeviceSynchronize();
    EXPECT_EQ(status, hipSuccess);
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
